#include "hip/hip_runtime.h"
#include "filter.h"
#include "cuda_timer.h"

#include <iostream>

using namespace std;


__global__
void kernel_sobel_filter(const uchar * input, uchar * output, const uint height, const uint width)
{
	// TODO
}

inline int divup(int a, int b)
{
	if (a % b)
		return a / b + 1;
	else
		return a / b;
}

/**
 * Wrapper for calling the kernel.
 */
void sobel_filter_gpu(const uchar * input, uchar * output, const uint height, const uint width)
{
	const int size = height * width * sizeof(uchar);

	CudaSynchronizedTimer timer;


	// Launch the kernel
	const int grid_x = 64;
	const int grid_y = 64;

	dim3 grid(1, 1, 1);  // TODO
	dim3 block(1, 1, 1); // TODO

	timer.start();
	kernel_sobel_filter<<<grid, block>>>(input, output, height, width);
	timer.stop();

	hipDeviceSynchronize();

	float time_kernel = timer.getElapsed();
}


void sobel_filter_cpu(const uchar * input, uchar * output, const uint height, const uint width)
{
	const int sobel_x[3][3] = {
		{-1, 0, 1},
		{-2, 0, 2},
		{-1, 0, 1}
	};
	const int sobel_y[3][3]  = {
		{-1, -2, -1},
		{0,   0,  0},
		{1,   2,  1}
	};

	for (uint y = 1; y < height - 1; ++y)
	{
		for (uint x = 1; x < width - 1; ++x)
		{

			const int pixel_x = (int) (
					(sobel_x[0][0] * input[x-1 + (y-1) * width]) + 
					(sobel_x[0][1] * input[x   + (y-1) * width]) + 
					(sobel_x[0][2] * input[x+1 + (y-1) * width]) +
					(sobel_x[1][0] * input[x-1 + (y  ) * width]) + 
					(sobel_x[1][1] * input[x   + (y  ) * width]) + 
					(sobel_x[1][2] * input[x+1 + (y  ) * width]) +
					(sobel_x[2][0] * input[x-1 + (y+1) * width]) + 
					(sobel_x[2][1] * input[x   + (y+1) * width]) + 
					(sobel_x[2][2] * input[x+1 + (y+1) * width])
					);
			const int pixel_y = (int) (
					(sobel_y[0][0] * input[x-1 + (y-1) * width]) + 
					(sobel_y[0][1] * input[x   + (y-1) * width]) + 
					(sobel_y[0][2] * input[x+1 + (y-1) * width]) +
					(sobel_y[1][0] * input[x-1 + (y  ) * width]) + 
					(sobel_y[1][1] * input[x   + (y  ) * width]) + 
					(sobel_y[1][2] * input[x+1 + (y  ) * width]) +
					(sobel_y[2][0] * input[x-1 + (y+1) * width]) + 
					(sobel_y[2][1] * input[x   + (y+1) * width]) + 
					(sobel_y[2][2] * input[x+1 + (y+1) * width])
					);

			float magnitude = sqrt((float)(pixel_x * pixel_x + pixel_y * pixel_y));

			if (magnitude < 0){ magnitude = 0; }
			if (magnitude > 255){ magnitude = 255; }

			output[x + y * width] = magnitude;
		}
	}
}



